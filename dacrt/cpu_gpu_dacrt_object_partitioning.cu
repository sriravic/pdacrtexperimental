#include "hip/hip_runtime.h"
#include <dacrt/dacrt.h>
#include <util/cutimer.h>

// Both kernels are defined in "bruteforcekernel.cu"
extern "C"
__global__ void segmentedBruteForce(RayArray rays, TriangleArray triangles, int* buffered_ray_ids, int ray_buffer_occupied, int* buffered_tri_ids, int tri_buffer_occupied,
	int* ray_segment_sizes, int* tri_segment_sizes, int* ray_segment_start, int* tri_segment_start, int num_segments, float* maxts,	int* hitids,
	int num_threads_launched, int num_blocks_launched);


extern "C" __global__ void updateMinKernel(int* ray_id, float* min_hits, int* minhit_ids, float* global_min, int* global_hits, int num_rays);

extern "C"
void dacrtCompleteRender(ParallelPack& pack, TriangleArray& dev_triangles, RayArray& dev_rays, DacrtRunTimeParameters& rtparams, Counters& ctr);

void cpuGpuDacrtObjectFunction(const AABB& space, 
	AabbArray& tri_aabbs,																							
	TriangleArray& triangles, TriangleArray& dev_triangles, int num_triangles, int* tri_idx_array, int tpivot,		
	RayArray& rays, RayArray& dev_rays, int num_rays, int* ray_idx_array, int rpivot,								
	ParallelPack& pack,
	float* maxts, int* hitids,																						
	DacrtRunTimeParameters& rtparams,
	Counters& ctr,
	Logger& logger
	) {

		/* Algorithm:
		=============
		// if ray_cnt | tr_cnt within their respective thresholds
				// check if we have enough space in buffer actually to do copy stuff
					// if YES just copy the bunch of values into the buffer
					// if NO, perform brute force of the entire buffer, segment wise
							// perform reduction operators for all rays having their ray ids {
							//		This might involve a sort operation followed by segmented reduction
							// }
					// check if after inserting the contents, we are within a threshold of the buffer
						// if yes, do brute force
						// else continue
		// else do triangle splitting
			// As of now, this operation is serial only ---> Heavily time intensive operation.
		*/

		if(tpivot < rtparams.PARALLEL_TRI_THRESHOLD || rpivot < rtparams.PARALLEL_RAY_THRESHOLD) {
			// we create a work queue and copy stuff?
BRUTE_FORCE:
#ifdef ENABLE_LOGGING
			logger.write(tpivot, rpivot);
#endif
			if((pack.ray_buffer_occupied + rpivot) < rtparams.BUFFER_SIZE && (pack.tri_buffer_occupied + tpivot) < rtparams.BUFFER_SIZE && pack.num_segments < rtparams.MAX_SEGMENTS) {
				ctr.raytri += tpivot * rpivot;
				Timer mem_cpy_timer("Memcpy Timer");
				mem_cpy_timer.start();
				thrust::copy(ray_idx_array, ray_idx_array + rpivot, pack.buffered_ray_idx.begin() + pack.ray_buffer_occupied);
				thrust::copy(tri_idx_array, tri_idx_array + tpivot, pack.buffered_tri_idx.begin() + pack.tri_buffer_occupied);
				pack.tri_segment_sizes[pack.num_segments] = tpivot;
				pack.ray_segment_sizes[pack.num_segments] = rpivot;
				pack.segment_ids[pack.num_segments] = pack.num_segments;
				pack.num_segments++;		// increment the count
				pack.ray_buffer_occupied += rpivot;
				pack.tri_buffer_occupied += tpivot;
				mem_cpy_timer.stop();
				ctr.mem_cpy_time += mem_cpy_timer.get_ms();
				//return;
			} else {
				// brute force parallel kernel
				thrust::device_vector<int> ray_segment_start(pack.num_segments);
				thrust::device_vector<int> tri_segment_start(pack.num_segments);
				thrust::exclusive_scan(pack.tri_segment_sizes.begin(), pack.tri_segment_sizes.begin() + pack.num_segments, tri_segment_start.begin());
				thrust::exclusive_scan(pack.ray_segment_sizes.begin(), pack.ray_segment_sizes.begin() + pack.num_segments, ray_segment_start.begin());
				
				// LOGIC1: We will launch one block for all segments. [very bad though]..
				int num_blocks = pack.num_segments;
				int num_threads_per_block = rtparams.NUM_RAYS_PER_BLOCK;
				
				Timer seg_brute_timer("SegmentedBruteForce Timer");
				
				seg_brute_timer.start();
				segmentedBruteForce<<<num_blocks, num_threads_per_block>>>(dev_rays, dev_triangles, thrust::raw_pointer_cast(&pack.buffered_ray_idx[0]), 
					pack.ray_buffer_occupied, thrust::raw_pointer_cast(&pack.buffered_tri_idx[0]), pack.tri_buffer_occupied, 
					thrust::raw_pointer_cast(&pack.ray_segment_sizes[0]), thrust::raw_pointer_cast(&pack.tri_segment_sizes[0]), 
					thrust::raw_pointer_cast(&ray_segment_start[0]), thrust::raw_pointer_cast(&tri_segment_start[0]), 
					pack.num_segments, thrust::raw_pointer_cast(&pack.buffered_ray_maxts[0]),
					thrust::raw_pointer_cast(&pack.buffered_ray_hitids[0]), num_threads_per_block * num_blocks, num_blocks);
				
				seg_brute_timer.stop();
				ctr.brute_force_time += seg_brute_timer.get_ms();
				// call kernel
				// now we can do a sort operation on the hitids, and maxts based on ray ids
				Timer seg_sort_timer("Seg Sorted Timer");
				seg_sort_timer.start();
				thrust::sort_by_key(pack.buffered_ray_idx.begin(), pack.buffered_ray_idx.begin() + pack.ray_buffer_occupied,
					thrust::make_zip_iterator(thrust::make_tuple(pack.buffered_ray_maxts.begin(), pack.buffered_ray_hitids.begin())));
				seg_sort_timer.stop();
				ctr.seg_sort_time += seg_sort_timer.get_ms();

				
				// now we have to reduce according to the key, which is the ray id
				static thrust::device_vector<int> ray_idx(rtparams.BUFFER_SIZE);
				static thrust::device_vector<float> ray_maxts(rtparams.BUFFER_SIZE);
				static thrust::device_vector<int> ray_hitids(rtparams.BUFFER_SIZE);
				static thrust::equal_to<int> pred;
				
				typedef thrust::device_vector<int>::iterator iter;
				typedef thrust::device_vector<float>::iterator fiter;
				typedef thrust::zip_iterator<thrust::tuple<fiter, iter> > zippy;
				thrust::pair<iter, zippy> minend;
				
				MinHitFunctor<thrust::tuple<float, int> > min_hit_functor;
				
				Timer reduction_timer("Reduction Timer");
				reduction_timer.start();
				minend = thrust::reduce_by_key(pack.buffered_ray_idx.begin(), pack.buffered_ray_idx.begin() + pack.ray_buffer_occupied,
					thrust::make_zip_iterator(thrust::make_tuple(pack.buffered_ray_maxts.begin(), pack.buffered_ray_hitids.begin())),
					ray_idx.begin(), thrust::make_zip_iterator(thrust::make_tuple(ray_maxts.begin(), ray_hitids.begin())),
					pred,
					min_hit_functor);
				reduction_timer.stop();
				ctr.reduction_time += reduction_timer.get_ms();
			
				// now we can update our global max_ts and hitid array
				int num_valid_keys = minend.first - ray_idx.begin();
				num_threads_per_block = 512;
				num_blocks = num_valid_keys / num_threads_per_block + (num_valid_keys % num_threads_per_block != 0);
				
				Timer update_min_timer("Update Min Timer");
				update_min_timer.start();
				updateMinKernel<<<num_blocks, num_threads_per_block>>>(thrust::raw_pointer_cast(&ray_idx[0]), thrust::raw_pointer_cast(&ray_maxts[0]), thrust::raw_pointer_cast(&ray_hitids[0]),
					thrust::raw_pointer_cast(&pack.dev_ray_maxts[0]), thrust::raw_pointer_cast(&pack.dev_hitids[0]), num_valid_keys);
				update_min_timer.stop();
				ctr.update_min_time += update_min_timer.get_ms();
				
				// reset the counters
				ray_idx.clear();
				ray_maxts.clear();
				ray_hitids.clear();
				pack.buffered_ray_idx.clear();
				pack.buffered_tri_idx.clear();
				pack.tri_segment_sizes.clear();
				pack.ray_segment_sizes.clear();
				pack.segment_ids.clear();
							
				pack.ray_buffer_occupied = 0;
				pack.tri_buffer_occupied = 0;
				pack.num_segments = 0;

				// now insert them into the buffer 
				// NOTE: this condition would have occured if we didnt have free space in the beginning itself. Now that we have cleared up space
				//       we can insert the original stuff we were supposed to insert.
				ctr.raytri += rpivot * tpivot;
				Timer mem_cpy_timer("Mem cpy Timer");
				mem_cpy_timer.start();
				thrust::copy(ray_idx_array, ray_idx_array + rpivot, pack.buffered_ray_idx.begin() + pack.ray_buffer_occupied);
				thrust::copy(tri_idx_array, tri_idx_array + tpivot, pack.buffered_tri_idx.begin() + pack.tri_buffer_occupied);
				pack.tri_segment_sizes[pack.num_segments] = tpivot;
				pack.ray_segment_sizes[pack.num_segments] = rpivot;
				pack.segment_ids[pack.num_segments]		= pack.num_segments;
				pack.num_segments++;
				pack.ray_buffer_occupied += rpivot;
				pack.tri_buffer_occupied += tpivot;
				mem_cpy_timer.stop();
				ctr.mem_cpy_time += mem_cpy_timer.get_ms();
				//return;
			}
			
		} else {
			
			float3 extents = space.bmax - space.bmin;
			{
			
				// left child
				int newtpivot;
				double start = omp_get_wtime();
				if(extents.x > extents.y && extents.x > extents.z) {
					float median = (space.bmax.x + space.bmin.x) * 0.5f;
					newtpivot = filterTrianglesObject(tri_aabbs, num_triangles, tri_idx_array, tpivot, median, splitTriangleLeftX, ctr.tribox);
				} else if(extents.y > extents.x && extents.y > extents.z) {
					float median = (space.bmax.y + space.bmin.y) * 0.5f;
					newtpivot = filterTrianglesObject(tri_aabbs, num_triangles, tri_idx_array, tpivot, median, splitTriangleLeftY, ctr.tribox);
				} else {
					float median = (space.bmax.z + space.bmin.z) * 0.5f;
					newtpivot = filterTrianglesObject(tri_aabbs, num_triangles, tri_idx_array, tpivot, median, splitTriangleLeftZ, ctr.tribox);
				}

				AABB left;
				for(int t = 0; t < newtpivot; t++) 
					left.unionWith(tri_aabbs.bmin[tri_idx_array[t]], tri_aabbs.bmax[tri_idx_array[t]]);		// note that we dont create that stupid intermediate AABB object
				
				double end = omp_get_wtime();
				ctr.tri_filter_time += static_cast<float>(end-start);

				start = omp_get_wtime();
				int newrpivot = filterRaysObject(rays, num_rays, ray_idx_array, left, rpivot, ctr.raybox);
				end = omp_get_wtime();
				ctr.ray_filter_time += static_cast<float>(end-start);

				if(newrpivot == rpivot || newtpivot == tpivot) {
					goto BRUTE_FORCE;
				} else {
					cpuGpuDacrtObjectFunction(left, tri_aabbs, triangles, dev_triangles, num_triangles, tri_idx_array, newtpivot, rays, dev_rays, num_rays, ray_idx_array,
						newrpivot, pack, maxts, hitids, rtparams, ctr, logger);
				}
			}
			{
				double start = omp_get_wtime();
				int newtpivot;
				if(extents.x > extents.y && extents.x > extents.z) {
					float median = (space.bmax.x + space.bmin.x) * 0.5f;
					newtpivot = filterTrianglesObject(tri_aabbs, num_triangles, tri_idx_array, tpivot, median, splitTriangleRightX, ctr.tribox);
				} else if(extents.y > extents.x && extents.y > extents.z) {
					float median = (space.bmax.y + space.bmin.y) * 0.5f;
					newtpivot = filterTrianglesObject(tri_aabbs, num_triangles, tri_idx_array, tpivot, median, splitTriangleRightY, ctr.raybox);
					
				} else {
					float median = (space.bmax.z + space.bmin.z) * 0.5f;
					newtpivot = filterTrianglesObject(tri_aabbs, num_triangles, tri_idx_array, tpivot, median, splitTriangleRightZ, ctr.raybox);
					
				}

				AABB right;
				for(int t = 0; t < newtpivot; t++) 
					right.unionWith(tri_aabbs.bmin[tri_idx_array[t]], tri_aabbs.bmax[tri_idx_array[t]]);		// note that we dont create that stupid intermediate AABB object
				
				double end = omp_get_wtime();
				ctr.tri_filter_time += static_cast<float>(end-start);

				start = omp_get_wtime();
				int newrpivot = filterRaysObject(rays, num_rays, ray_idx_array, right, rpivot, ctr.raybox);
				end = omp_get_wtime();
				ctr.ray_filter_time += static_cast<float>(end-start);

				if(newrpivot == rpivot || newtpivot == tpivot) {
					goto BRUTE_FORCE;
				} else {
					cpuGpuDacrtObjectFunction(right, tri_aabbs, triangles, dev_triangles, num_triangles, tri_idx_array, newtpivot, rays, dev_rays, num_rays, ray_idx_array, newrpivot,
						pack, maxts, hitids, rtparams, ctr, logger);
				}
			}
		}
}


void cpuGpuDacrtObjectPartitioning(const AABB& sceneBox, 
	TriangleArray& triangles, AabbArray& tri_aabbs, RayArray& rays,
	TriangleArray& dev_triangles, RayArray& dev_rays,
	int num_triangles, int* tri_idx_array, int tpivot, 
	int num_rays, int* ray_idx_array, int rpivot,															
	float* maxts, int* hitids,
	DacrtRunTimeParameters& rtparams,
	Counters& ctr,
	Logger& logger
	) {

		thrust::device_vector<int>		buffered_ray_idx(rtparams.BUFFER_SIZE);
		thrust::device_vector<int>		buffered_tri_idx(rtparams.BUFFER_SIZE);
		thrust::device_vector<int>		segment_ids(rtparams.MAX_SEGMENTS);
		thrust::device_vector<int>		ray_segment_sizes(rtparams.MAX_SEGMENTS);
		thrust::device_vector<int>		tri_segment_sizes(rtparams.MAX_SEGMENTS);
		thrust::device_vector<float>	buffered_ray_maxts(rtparams.BUFFER_SIZE, FLT_MAX);
		thrust::device_vector<int>		buffered_ray_hitids(rtparams.BUFFER_SIZE, -1);
		thrust::device_vector<float>	dev_ray_maxts(num_rays, FLT_MAX);
		thrust::device_vector<int>		dev_hitids(num_rays, -1);


		int ray_buffer_occupied = 0;
		int tri_buffer_occupied = 0;
		int num_segments = 0;

		ParallelPack pack(buffered_ray_idx, buffered_tri_idx, segment_ids, ray_segment_sizes, tri_segment_sizes, buffered_ray_maxts, buffered_ray_hitids,
			dev_ray_maxts, dev_hitids, ray_buffer_occupied, tri_buffer_occupied, num_segments);

		printf("Starting dacrt in parallel\n");

		cpuGpuDacrtObjectFunction(sceneBox, tri_aabbs, triangles, dev_triangles, num_triangles, tri_idx_array, tpivot, rays, dev_rays, num_rays, ray_idx_array, rpivot,
			pack, maxts, hitids, rtparams, ctr, logger);

		// complete the incomplete segments
		// now we might have come out with a half full buffer. We just have to complete that also.
		if(pack.num_segments > 0) 
			dacrtCompleteRender(pack, dev_triangles, dev_rays, rtparams, ctr);
		
		// now pack.dev_ray_maxts will have all the required values
		// and the pack.dev_ray_hitids will have hitids.
		// copy them into host buffer for rendering
		thrust::copy(dev_ray_maxts.begin(), dev_ray_maxts.end(), maxts);
		thrust::copy(dev_hitids.begin(), dev_hitids.end(), hitids);
}



